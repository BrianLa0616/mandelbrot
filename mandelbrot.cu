#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include "stb_image_write.h"

// Include STB library implementation
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

// CUDA kernel for Mandelbrot set calculation
__device__ int mandelbrot(double real, double imag) {
    int limit = 10000;
    double zReal = real;
    double zImag = imag;

    for (int i = 0; i < limit; ++i) {
        double r2 = zReal * zReal;
        double i2 = zImag * zImag;

        if (r2 + i2 > 4.0)
            return i;

        zImag = 2.0 * zReal * zImag + imag;
        zReal = r2 - i2 + real;
    }
    return limit;
}

// CUDA kernel for pixel calculation
__global__ void calculatePixels(unsigned char* image, int width, int height, 
                              double x_start, double x_fin, double y_start, double y_fin) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < width && idy < height) {
        double dx = (x_fin - x_start) / (width - 1);
        double dy = (y_fin - y_start) / (height - 1);

        double x = x_start + idx * dx;
        double y = y_fin - idy * dy;

        int value = mandelbrot(x, y);

        // Color mapping
        unsigned char r, g, b;
        if (value == 100) {
            r = g = b = 0; // Black
        } else if (value > 90) {
            r = 139; g = 0; b = 0; // Dark Red
        } else if (value > 70) {
            r = 255; g = 0; b = 0; // Bright Red
        } else if (value > 50) {
            r = 255; g = 165; b = 0; // Orange
        } else if (value > 30) {
            r = 255; g = 255; b = 0; // Yellow
        } else if (value > 20) {
            r = 0; g = 255; b = 0; // Green
        } else if (value > 10) {
            r = 0; g = 255; b = 255; // Cyan
        } else if (value > 5) {
            r = 0; g = 0; b = 255; // Blue
        } else if (value > 3) {
            r = 128; g = 0; b = 128; // Purple
        } else {
            r = 255; g = 105; b = 180; // Hot Pink
        }

        // Set pixel color in the image
        int pixelIndex = (idy * width + idx) * 3;
        image[pixelIndex] = r;
        image[pixelIndex + 1] = g;
        image[pixelIndex + 2] = b;
    }
}

int main() {
    const int width = 1000;
    const int height = 800;
    const size_t imageSize = width * height * 3 * sizeof(unsigned char);

    // Host memory
    unsigned char* h_image = (unsigned char*)malloc(imageSize);

    // Device memory
    unsigned char* d_image;
    hipMalloc(&d_image, imageSize);

    // Set computation parameters
    double x_start = -2.0;
    double x_fin = 1.0;
    double y_start = -1.0;
    double y_fin = 1.0;

    // Define grid and block dimensions
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                  (height + blockSize.y - 1) / blockSize.y);

    // Launch kernel
    calculatePixels<<<gridSize, blockSize>>>(d_image, width, height,
                                           x_start, x_fin, y_start, y_fin);

    // Check for kernel launch errors
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }

    // Copy result back to host
    hipMemcpy(h_image, d_image, imageSize, hipMemcpyDeviceToHost);

    // Write image to file
    stbi_write_png("mandelbrot_cuda.png", width, height, 3, h_image, width * 3);
    printf("Mandelbrot set image saved as mandelbrot_cuda.png\n");

    // Cleanup
    hipFree(d_image);
    free(h_image);

    return 0;
}